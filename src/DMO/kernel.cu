#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <cstdint>
#include <stdio.h>
#include <cfloat>

#include <type_traits>

#include "hip/hip_math_constants.h"

#include "Vertex.h"
#include "DmoParams.h"
#include "DmoMesh.h"
#include "gpuErrchk.h"
#include "DMO_PUBLIC.h"

namespace DMO
{

	typedef union
	{
		float floats[2];             // floats[0] = lowest
		std::int32_t ints[2];        // ints[1] = lowIdx
		unsigned long long ulong;    // for atomic update
	} floatIntUnion;

	__device__ unsigned long long floatIntArgMax( unsigned long long* address, float val, std::int32_t idx ) {
		floatIntUnion loc, newValue;
		loc.floats[0] = val;
		loc.ints[1] = idx;
		newValue.ulong = *address;
		while( newValue.floats[0] < val )
			newValue.ulong = atomicCAS( address, newValue.ulong, loc.ulong );

		return newValue.ulong;
	}

	template<typename MetricT>
	__global__ void optimizeHierarchical( float2* points, const int* coloredVertexIDs, const int cOff, const DmoVertex* vertices, const int* oneRingVec, const MetricT metric ) {

		const uint2 idx1 = {
			threadIdx.x / NQ,
			threadIdx.x % NQ
		};
		const uint2 idx2 = {
			( threadIdx.x + NQ * NQ / 2 ) / NQ,
			( threadIdx.x + NQ * NQ / 2 ) % NQ
		};

		const DmoVertex& v = vertices[coloredVertexIDs[cOff + blockIdx.x]];

		float q = -FLT_MAX;

		__shared__ float2 currPos;	// current optimal position
		__shared__ float2 maxDist;

		__shared__ floatIntUnion argMaxVal;
		argMaxVal.floats[0] = -FLT_MAX;
		argMaxVal.ints[1] = NQ * NQ;

		__shared__ float2 oneRing[MAX_ONE_RING_SIZE];

		// min/max search + loading oneRing
		if( threadIdx.x == 0 ) {
			maxDist.x = -FLT_MAX;
			maxDist.y = -FLT_MAX;

			for( int k = 0; k < v.oneRingSize - 1; ++k ) {
				float2 vo = points[oneRingVec[v.oneRingID + k]];
				oneRing[k] = vo;

				float2 dist = {
					abs( points[v.idx].x - vo.x ),
					abs( points[v.idx].y - vo.y )
				};

				maxDist.x = fmaxf( maxDist.x, dist.x );
				maxDist.y = fmaxf( maxDist.y, dist.y );
			}

			oneRing[v.oneRingSize - 1] = points[oneRingVec[v.oneRingID + v.oneRingSize - 1]];

			currPos = points[v.idx];
		}

		// start depth iteration
		float depth_scale = GRID_SCALE;
		for( int depth = 0; depth < DEPTH; ++depth ) {

			float2 aabbMin, aabbMax;	// axis aligned bounding box
			aabbMin.x = currPos.x - depth_scale * maxDist.x;
			aabbMin.y = currPos.y - depth_scale * maxDist.y;
			aabbMax.x = currPos.x + depth_scale * maxDist.x;
			aabbMax.y = currPos.y + depth_scale * maxDist.y;

			float2 p1 = {
				AFFINE_FACTOR * ( idx1.x * aabbMin.x + ( NQ - 1 - idx1.x ) * aabbMax.x ),
				AFFINE_FACTOR * ( idx1.y * aabbMin.y + ( NQ - 1 - idx1.y ) * aabbMax.y )
			};
			float2 p2 = {
				AFFINE_FACTOR * ( idx2.x * aabbMin.x + ( NQ - 1 - idx2.x ) * aabbMax.x ),
				AFFINE_FACTOR * ( idx2.y * aabbMin.y + ( NQ - 1 - idx2.y ) * aabbMax.y )
			};

			float q1 = metric.vertexQuality( oneRing, v.oneRingSize, points[v.idx], p1 );
			float q2 = metric.vertexQuality( oneRing, v.oneRingSize, points[v.idx], p2 );
			//float q1 = metric.vertexQuality( oneRing, v.oneRingSize, p1 );
			//float q2 = metric.vertexQuality( oneRing, v.oneRingSize, p2 );

			float argMax = 0;
			if( q1 > q2 ) {
				q = q1;
				argMax = 1;
			} else {
				q = q2;
				argMax = 2;
			}

			__syncwarp();
			//__syncthreads();
			floatIntArgMax( ( unsigned long long* ) & ( argMaxVal.ulong ), q, idx1.x * NQ + idx1.y );

			float qOld = metric.vertexQuality( oneRing, v.oneRingSize, points[v.idx], currPos );
			//float qOld = metric.vertexQuality( oneRing, v.oneRingSize, currPos );
			if( idx1.x * NQ + idx1.y == argMaxVal.ints[1] && qOld < q ) {
				if( argMax == 1 ) {
					currPos.x = p1.x;
					currPos.y = p1.y;
				} else {
					currPos.x = p2.x;
					currPos.y = p2.y;
				}
			}

			//rescale candidate grid to the size of two cells
			depth_scale *= 2 * AFFINE_FACTOR;
		}

		// set new position if it is better than the old one
		float qOld = metric.vertexQuality( oneRing, v.oneRingSize, points[v.idx], points[v.idx] );
		//float qOld = metric.vertexQuality( oneRing, v.oneRingSize, points[v.idx] );
		if( idx1.x * NQ + idx1.y == argMaxVal.ints[1] && qOld < q ) {
			points[v.idx].x = currPos.x;
			points[v.idx].y = currPos.y;
		}
	}

	template<typename MetricT>
	void DMO_PUBLIC dmoGPU( thrust::device_vector<float2>& points_d, DmoMesh& dmoMesh1_, const MetricT& metric ) {
		for( auto colorIt = dmoMesh1_.colorOffset_.begin(); colorIt != dmoMesh1_.colorOffset_.end() - 1; ++colorIt ) {
			const int nBlocks = *( colorIt + 1 ) - *colorIt;
			optimizeHierarchical << <nBlocks, N_THREADS >> > (
				points_d.data().get(),
				dmoMesh1_.coloredVertexIDs_d.data().get(),
				*colorIt,
				dmoMesh1_.vertices_d.data().get(),
				dmoMesh1_.oneRingVec_d.data().get(),
				metric
				);
			gpuErrchk( hipDeviceSynchronize() );
		}
	}
}

#define REGISTER_METRIC(metric) \
namespace DMO { template void DMO_PUBLIC dmoGPU<>( thrust::device_vector<float2>&, DmoMesh&, const metric& ); }

#include "Metrics.h"