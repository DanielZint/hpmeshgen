#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <cstdint>
#include <stdio.h>
#include <cfloat>

#include <type_traits>

#include "hip/hip_math_constants.h"

//#include "../BackgroundGrid/SizeGrid.h"
#include "Stopwatch/Stopwatch.h"
// Open Mesh
#include "MeshHeader.h"

#define Q_MEANRATIO 0
#define Q_JACOBIAN 4
#define Q_MINANGLE 5
#define Q_RADIUSRATIO 6
#define Q_MAXANGLE 7
#define Q_CONDITION 8
#define Q_COMBLAPLACEMEANRATIO 9


/* Keep NQ = 8 for two dimensional meshes! This value was chosen because it gives optimal
performance considering a warp-size of 32 because NQ = 8 results in 8 * 8 = 64 nodes
which is double the warp size. Each vertex is computed using one warp where each warp
computes two grid nodes.
Another implementation used 2 warps for one grid but it was slower as syncthreads is
too expensive.
*/
// Size of Quality Mesh
constexpr int NQ = 8;
// number of refinement steps within DMO
constexpr int DEPTH = 3;
// double the maximal number of allowed vertices on the one-ring neighborhood
constexpr int MAX_ONE_RING_SIZE = 32;

// For quality output
constexpr int N_QUALITY_COLS = 10;
// Set this value to print quality
#define PRINT_QUALITY 0


// Error output
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		//fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		fprintf(stderr, "GPUassert: %s. Line %d\n", hipGetErrorString(code), line);
		if (abort) exit(code);
	}
}


typedef union {
	float floats[2];                 // floats[0] = lowest
	std::int32_t ints[2];                     // ints[1] = lowIdx
	unsigned long long ulong;    // for atomic update
} my_atomics;

__device__ unsigned long long my_atomicArgMax(unsigned long long* address, float val, std::int32_t idx)
{
	my_atomics loc, newValue;
	loc.floats[0] = val;
	loc.ints[1] = idx;
	newValue.ulong = *address;
	while (newValue.floats[0] < val)
		newValue.ulong = atomicCAS(address, newValue.ulong, loc.ulong);
		
	return newValue.ulong;
}


/*
Holds information of the vertex-id, the number of neighbors, and their location in the one-ring vector.
*/
struct Vertex {
	int oneRingID;
	int n_oneRing;
	int id;				// own vertex id
};

/////////////////////////////////
// quality metrics per element //
__host__ __device__ __forceinline__ float meanRatioTri(const float p[3][2]) {

	float e[3][2];
	float e_length_squared[3];

	for (int i = 0; i < 3; ++i) {
		int j = (i + 1) % 3;
		e[i][0] = p[j][0] - p[i][0];
		e[i][1] = p[j][1] - p[i][1];

		e_length_squared[i] = e[i][0] * e[i][0] + e[i][1] * e[i][1];
	}

	float l = e_length_squared[0] + e_length_squared[1] + e_length_squared[2];
	float area = e[0][0] * e[1][1] - e[0][1] * e[1][0];

	if( area < 0 )
		return area;
	return 2.f * sqrt(3.f) * area / l;
}

__host__ __device__ __forceinline__ float conditionQuad( const float p[4][2] ) {

	float e[4][2];
	float els[4];

	for( int i = 0; i < 4; ++i ) {
		int j = ( i + 1 ) % 4;
		e[i][0] = p[j][0] - p[i][0];
		e[i][1] = p[j][1] - p[i][1];
		els[i] = e[i][0] * e[i][0] + e[i][1] * e[i][1];
	}
	
	auto detJ1 = e[0][1] * e[3][0] - e[0][0] * e[3][1];
	auto l1 = els[0] + els[3];
	auto q1 = 2 * detJ1 / l1;

	auto detJ2 = e[1][1] * e[0][0] - e[1][0] * e[0][1];
	auto l2 = els[1] + els[0];
	auto q2 = 2 * detJ2 / l2;

	//auto detJ3 = e[2][1] * e[1][0] - e[2][0] * e[1][1];
	//auto l3 = e_length_squared[2] + e_length_squared[1];
	//auto q3 = 2 * detJ3 / l3;

	auto detJ4 = e[3][1] * e[2][0] - e[3][0] * e[2][1];
	auto l4 = els[3] + els[2];
	auto q4 = 2 * detJ4 / l4;

	auto detMin = fminf( detJ1, detJ2 );
	detMin = fminf( detMin, detJ4 );

	if( detMin < 0 )
		return detMin;

	auto qMin = fminf( q1, q2 );
	qMin = fminf( qMin, q4 );

	return qMin;
}
/////////////////////////////////

///////////////////////////////////////////
// distinguish between triangle and quad //

__host__ __device__ __forceinline__ float computeShapeQuality(const int n_oneRing, const float oneRing[MAX_ONE_RING_SIZE], const float p[2], const int element_size) {
	float q = FLT_MAX;

	switch (element_size)
	{
	case 3:
		for (int k = 0; k < n_oneRing - 1; ++k) {
			float v[3][2] = { { p[0], p[1] },{ oneRing[2 * k], oneRing[2 * k + 1] },{ oneRing[2 * (k + 1)], oneRing[2 * (k + 1) + 1] } };
			q = fminf(q, meanRatioTri(v));
		}
		break;
	default:
		break;
	}

	return q;
}

__host__ __device__ __forceinline__ float computeConditionQuality( const int n_oneRing, const float oneRing[MAX_ONE_RING_SIZE], const float p[2], const int element_size ) {
	float q = FLT_MAX;

	switch( element_size ) {
	case 3:
		break;
	case 4:
		for( int k = 0; k < n_oneRing - 1; k += 2 ) {
			float v[4][2] = { { p[0], p[1] },{ oneRing[2 * k], oneRing[2 * k + 1] },{ oneRing[2 * ( k + 1 )], oneRing[2 * ( k + 1 ) + 1] },{ oneRing[2 * ( k + 2 )], oneRing[2 * ( k + 2 ) + 1] } };
			q = fminf( q, conditionQuad( v ) );
		}
		break;
	default:
		break;
	}

	return q;
}

__host__ __device__ __forceinline__ float computeCombinedLaplaceMeanRatioQuality( const int n_oneRing, const float oneRing[MAX_ONE_RING_SIZE], const float p[2], const int element_size ) {
	float q = FLT_MAX;

	float lp[2] = { 0,0 };
	
	switch( element_size ) {
	case 3:
		// compute laplace point
		for( int k = 0; k < n_oneRing - 1; ++k ) {
			lp[0] += oneRing[2 * k];
			lp[1] += oneRing[2 * k + 1];
		}
		lp[0] /= ( n_oneRing - 1 );
		lp[1] /= ( n_oneRing - 1 );
		lp[0] = p[0] - lp[0];
		lp[1] = p[1] - lp[1];

		for( int k = 0; k < n_oneRing - 1; ++k ) {
			float v[3][2] = { { p[0], p[1] },{ oneRing[2 * k], oneRing[2 * k + 1] },{ oneRing[2 * ( k + 1 )], oneRing[2 * ( k + 1 ) + 1] } };
			q = fminf( q, meanRatioTri( v ) );
		}
		break;
	case 4:
	{
		printf( "This metric is not implemented for triangles yet.\n" );
		break;
	}
	default:
		break;
	}

	if( q < 0.5 )
		return q;
	else
		return 0.5f + 1.f / ( lp[0] * lp[0] + lp[1] * lp[1] + 1 );
}

__host__ __device__ __forceinline__ float quality(const int n_oneRing, const float oneRing[MAX_ONE_RING_SIZE], const float p[2], const int element_size, const int q_crit) {
	switch (q_crit)
	{
	case Q_MEANRATIO:
		return computeShapeQuality(n_oneRing, oneRing, p, element_size);
	case Q_CONDITION:
		return computeConditionQuality( n_oneRing, oneRing, p, element_size );
	case Q_COMBLAPLACEMEANRATIO:
		return computeCombinedLaplaceMeanRatioQuality( n_oneRing, oneRing, p, element_size );
	default:
		return -1;
	}
}


__global__ void printFaceQuality(const float* points, const int* faceVec, const int n_faces, const int element_size, const int q_crit) {
	static int counter = 0;


	int q_vec[N_QUALITY_COLS] = { 0 };
	float q_min = FLT_MAX;

	for (int i = 0; i < n_faces; ++i) {
		float q;

		switch (q_crit)
		{
		case Q_MEANRATIO:
			switch (element_size)
			{
			case 3:
			{
				int v_id[3] = { faceVec[3 * i],  faceVec[3 * i + 1], faceVec[3 * i + 2] };
				float p[3][2];
				for (int i = 0; i < 3; ++i) {
					p[i][0] = points[2 * v_id[i]];
					p[i][1] = points[2 * v_id[i] + 1];
				}

				q = meanRatioTri(p);
				break;
			}
			default:
				printf("Quality for this type of element is unknown. Element-size = %d\n", element_size);
				return;
			}
			break;
		default:
			printf("Quality metric unknown\n");
			break;
		}

		q_vec[int(q * N_QUALITY_COLS - 0.0001)] += 1;

		q_min = fminf(q_min, q);
	}

	printf("%3d: ", counter++);

	for (int i = 0; i < N_QUALITY_COLS; ++i) {
		if (q_vec[i] != 0)
			printf("%4d | ", q_vec[i]);
		else
			printf("     | ");
	}
	printf("| q_min = %1.6f", q_min);
	printf("\n");

}

__global__ void printFaceQuality(const float* points, const int* faceVec, const int n_faces, const int element_size, const int q_crit, float* q_min_vec, float* q_avg_vec) {
	static int counter = 0;

	float q_min = FLT_MAX;
	float q_avg = 0;

	for (int i = 0; i < n_faces; ++i) {
		float q;

		switch (q_crit)
		{
		case Q_MEANRATIO:
			switch (element_size)
			{
			case 3:
			{
				int v_id[3] = { faceVec[3 * i],  faceVec[3 * i + 1], faceVec[3 * i + 2] };
				float p[3][2];
				for (int i = 0; i < 3; ++i) {
					p[i][0] = points[2 * v_id[i]];
					p[i][1] = points[2 * v_id[i] + 1];
				}

				q = meanRatioTri(p);
				break;
			}
			default:
				printf("Quality for this type of element is unknown. Element-size = %d\n", element_size);
				return;
			}
			break;
		default:
			printf("Quality metric unknown\n");
			break;
		}

		q_min = fminf(q_min, q);
		q_avg += q;
	}
	q_avg /= n_faces;

	q_min_vec[counter] = q_min;
	q_avg_vec[counter++] = q_avg;
}


__global__ void optimizeHierarchical(int* coloredVertexIDs, const int cOff, const Vertex* vertices, float* points, int* oneRingVec, const float affineFactor_, const int element_size, const int q_crit, const float grid_scale) {
	const int i1 = threadIdx.x / NQ;
	const int j1 = threadIdx.x % NQ;

	const int i2 = (threadIdx.x + NQ * NQ / 2) / NQ;
	const int j2 = (threadIdx.x + NQ * NQ / 2) % NQ;

	const Vertex& v = vertices[coloredVertexIDs[cOff + blockIdx.x]];

	float q = -FLT_MAX;

	__shared__ float xPos, yPos;
	__shared__ float maxDistx;
	__shared__ float maxDisty;

	__shared__ my_atomics argMaxVal;
	argMaxVal.floats[0] = -FLT_MAX;
	argMaxVal.ints[1] = NQ*NQ;

	__shared__ float oneRing[MAX_ONE_RING_SIZE];

	// min/max search + loading oneRing
	if (threadIdx.x == 0) {
		maxDistx = -FLT_MAX;
		maxDisty = -FLT_MAX;

		for (int k = 0; k < v.n_oneRing - 1; ++k) {
			float oneRingX = points[2 * oneRingVec[v.oneRingID + k]];
			float oneRingY = points[2 * oneRingVec[v.oneRingID + k] + 1];
			oneRing[2 * k] = oneRingX;
			oneRing[2 * k + 1] = oneRingY;

			float xDist = abs(points[2 * v.id] - oneRingX);
			float yDist = abs(points[2 * v.id + 1] - oneRingY);

			maxDistx = fmaxf(maxDistx, xDist);
			maxDisty = fmaxf(maxDisty, yDist);
		}
		
		oneRing[2 * v.n_oneRing - 2] = points[2 * oneRingVec[v.oneRingID + v.n_oneRing - 1]];
		oneRing[2 * v.n_oneRing - 1] = points[2 * oneRingVec[v.oneRingID + v.n_oneRing - 1] + 1];

		xPos = points[2 * v.id];
		yPos = points[2 * v.id + 1];
	}

	// special case of valence 2 in a quad mesh
	if( element_size == 4 && v.n_oneRing == 5 ) {
		if( threadIdx.x == 0 ) {
			auto x1 = oneRing[2 * 0];
			auto y1 = oneRing[2 * 0 + 1];
			auto x2 = oneRing[2 * 2];
			auto y2 = oneRing[2 * 2 + 1];
			xPos = 0.5f * ( x1 + x2 );
			yPos = 0.5f * ( y1 + y2 );
			points[2 * v.id] = xPos;
			points[2 * v.id + 1] = yPos;
		}
		return;
	}

	// start depth iteration
	float depth_scale = grid_scale;
	float argMax = 0;
	for (int depth = 0; depth < DEPTH; ++depth) {

		float xMax, xMin, yMax, yMin;
		xMax = xPos + depth_scale * maxDistx;
		xMin = xPos - depth_scale * maxDistx;
		yMax = yPos + depth_scale * maxDisty;
		yMin = yPos - depth_scale * maxDisty;


		float pos_i1 = affineFactor_ * (i1 * xMin + (NQ - 1 - i1) * xMax);
		float pos_j1 = affineFactor_ * (j1 * yMin + (NQ - 1 - j1) * yMax);
		float pos_i2 = affineFactor_ * (i2 * xMin + (NQ - 1 - i2) * xMax);
		float pos_j2 = affineFactor_ * (j2 * yMin + (NQ - 1 - j2) * yMax);

		float p1[2] = { pos_i1, pos_j1 };
		float q1 = quality(v.n_oneRing, oneRing, p1, element_size, q_crit);
		float p2[2] = { pos_i2, pos_j2 };
		float q2 = quality(v.n_oneRing, oneRing, p2, element_size, q_crit);

		if (q1 > q2) {
			q = q1;
			argMax = 1;
		}
		else {
			q = q2;
			argMax = 2;
		}

		my_atomicArgMax( (unsigned long long *)&( argMaxVal.ulong ), q, i1 * NQ + j1 );

		float pCurrent[2] = { xPos, yPos };
		float qOld = quality(v.n_oneRing, oneRing, pCurrent, element_size, q_crit);
		if (i1 * NQ + j1 == argMaxVal.ints[1] && qOld < q) {
			if (argMax == 1) {
				xPos = pos_i1;
				yPos = pos_j1;
			}
			else {
				xPos = pos_i2;
				yPos = pos_j2;
			}
		}
		
		//depth dependent scaling factor
		depth_scale = depth_scale * (2.f / (NQ - 1));
	}

	// set new position if it is better than the old one
	float pOld[2] = { points[2 * v.id] , points[2 * v.id + 1] };
	float qOld = quality( v.n_oneRing, oneRing, pOld, element_size, q_crit );
	if (i1 * NQ + j1 == argMaxVal.ints[1] && qOld < q) {
		points[2 * v.id] = xPos;
		points[2 * v.id + 1] = yPos;
	}
}


struct UniformGrid
{
	int nx, ny;
	float hx, hy, xMin, yMin, xMax, yMax;
};


///////////////////////////////////////////////////////////////////////////////
/////////////////////////////// OpenMesh //////////////////////////////////////

template<typename T> inline void copyOpenMeshData(T& mesh, float* points, Vertex* vertices, int* oneRingVec) {

	bool isPolyMesh = std::is_same<T, PolyMesh>::value;

	int interior_counter = 0;
	int oneRing_counter = 0;
	for (auto v_it = mesh.vertices_begin(); v_it != mesh.vertices_end(); ++v_it) {
		TriMesh::Point p = mesh.point(*v_it);

		points[2 * v_it->idx()] = p[0];
		points[2 * v_it->idx() + 1] = p[1];


		if (!mesh.is_boundary(*v_it)) {
			// fill vertex struct

			Vertex& v = vertices[interior_counter];
			v.id = v_it->idx();

			v.n_oneRing = 0;
			for (auto voh_it = mesh.voh_iter(*v_it); voh_it.is_valid(); ++voh_it) {
				++v.n_oneRing;
				if (isPolyMesh && !mesh.is_boundary(*voh_it)) ++v.n_oneRing;
			}
			++v.n_oneRing;

			v.oneRingID = oneRing_counter;

			TriMesh::HalfedgeHandle heh = *(mesh.voh_iter(*v_it));
			TriMesh::HalfedgeHandle heh_init = heh;

			do {
				oneRingVec[oneRing_counter++] = mesh.to_vertex_handle(heh).idx();
				if (isPolyMesh) {
					heh = mesh.next_halfedge_handle(heh);
					oneRingVec[oneRing_counter++] = mesh.to_vertex_handle(heh).idx();
				}
				heh = mesh.next_halfedge_handle(heh);
				heh = mesh.next_halfedge_handle(heh);
				heh = mesh.opposite_halfedge_handle(heh);
			} while (heh.idx() != heh_init.idx());

			oneRingVec[oneRing_counter] = mesh.to_vertex_handle(heh).idx();
			++oneRing_counter;

			++interior_counter;
		}
	}

}

template <typename T> inline void createColoring(T& mesh, const int n_free_vertices, int** coloredVertexIDs, std::vector<int>& colorOffset_) {

	bool isPolyMesh = std::is_same<T, PolyMesh>::value;

	// create coloring scheme
	std::vector<int>colorScheme(mesh.n_vertices(), -1);
	int colorSchemeIt = 0;

	// set boundarys to a value that can be ignored
	for (auto v_it = mesh.vertices_begin(); v_it != mesh.vertices_end(); ++v_it) {
		if (mesh.is_boundary(*v_it)) {
			colorScheme[v_it->idx()] = -2;
		}
	}

	while (std::find(colorScheme.begin(), colorScheme.end(), -1) != colorScheme.end()) {
		for (auto v_it = mesh.vertices_begin(); v_it != mesh.vertices_end(); ++v_it) {
	
			if (colorScheme[v_it->idx()] != -1) { continue; }		// vertex is already colored
	
			bool neighborIsCurrent = false;
			for (auto voh_it = mesh.voh_iter(*v_it); voh_it.is_valid(); ++voh_it) {
				PolyMesh::VertexHandle vh1 = mesh.to_vertex_handle(*voh_it);
				if (!isPolyMesh && colorScheme[vh1.idx()] == colorSchemeIt) {
					neighborIsCurrent = true;
					break;
				}
				else if (isPolyMesh) {
					PolyMesh::VertexHandle vh2 = mesh.to_vertex_handle(mesh.next_halfedge_handle(*voh_it));
					if (colorScheme[vh1.idx()] == colorSchemeIt || colorScheme[vh2.idx()] == colorSchemeIt) {
						neighborIsCurrent = true;
						break;
					}
				}
			}
			if (neighborIsCurrent) { continue; }			// a neighboring vertex is already in this color
	
			colorScheme[v_it->idx()] = colorSchemeIt;
		}
		++colorSchemeIt;
	}

	//// DEBUG one color for each vertex
	//for ( auto v_it = mesh.vertices_begin(); v_it != mesh.vertices_end(); ++v_it ) {
	//	if ( colorScheme[v_it->idx()] == -1 ) {
	//		colorScheme[v_it->idx()] = colorSchemeIt++;
	//	}
	//}

	int n_colors = *(std::max_element(colorScheme.begin(), colorScheme.end())) + 1;

	if( n_colors == -1 )
		return;

	std::vector<int> n_color_vecs(n_colors, 0);
	for (int i = 0; i < colorScheme.size(); ++i) {
		if (colorScheme[i] > -1)
			++n_color_vecs[colorScheme[i]];
	}

	*coloredVertexIDs = new int[n_free_vertices];

	colorOffset_ = std::vector<int>(n_colors + 1, 0);
	for (int i = 1; i < n_colors; ++i) {
		colorOffset_[i] = colorOffset_[i - 1] + n_color_vecs[i - 1];
	}
	colorOffset_[n_colors] = n_free_vertices;		// mark the end of the colored-vertices vector

													// add vertex ids
	std::vector<int>colorCounter(n_colors, 0);
	int interior_counter = 0;
	for (int i = 0; i < colorScheme.size(); ++i) {
		if (colorScheme[i] < 0) { continue; }
		(*coloredVertexIDs)[colorOffset_[colorScheme[i]] + colorCounter[colorScheme[i]]++] = interior_counter++;
	}
}

// DEBUG VARIABLES //
int runCounter = 0;
float* vertexPosBuf;

template <typename T> void discreteMeshOptimization(T& mesh, const int q_crit = Q_MEANRATIO, const float grid_scale = 0.5f, int n_iter = 100) {

	constexpr bool isPolyMesh = std::is_same<T, PolyMesh>::value;
	constexpr int element_size = std::is_same<T, PolyMesh>::value ? 4 : 3;

	Stopwatch sw;
	int n_free_vertices = 0;
	int oneRingVecLength = 0;
#pragma omp parallel for reduction(+:n_free_vertices,oneRingVecLength)
	for (int i = 0; i < mesh.n_vertices(); ++i) {
		PolyMesh::VertexHandle vh = mesh.vertex_handle(i);
		if (mesh.is_boundary(vh)) { continue; }
		++n_free_vertices;

		for (auto voh_it = mesh.voh_iter(vh); voh_it.is_valid(); ++voh_it) {
			++oneRingVecLength;
			if (isPolyMesh && !mesh.is_boundary(*voh_it)) ++oneRingVecLength;
		}
		++oneRingVecLength;		// additional count s.th. last element is again the first element
	}

	if( n_free_vertices == 0 ) {
		return;
	}

	// convert OpenMesh to a basic structure
	float* points = new float[2 * mesh.n_vertices()];
	Vertex* vertices = new Vertex[n_free_vertices];
	int* oneRingVec = new int[oneRingVecLength];

	float* points_d;
	Vertex* vertices_d;
	int* oneRingVec_d;
	int* coloredVertexIDs_d;

	int* coloredVertexIDs;
	std::vector<int> colorOffset_;


#pragma omp parallel sections num_threads(2)
	{
#pragma omp section
		{
			gpuErrchk(hipMalloc((void**)&points_d, 2 * mesh.n_vertices() * sizeof(float)));
			gpuErrchk(hipMalloc((void**)&vertices_d, n_free_vertices * sizeof(Vertex)));
			gpuErrchk(hipMalloc((void**)&oneRingVec_d, oneRingVecLength * sizeof(int)));
			gpuErrchk(hipMalloc((void**)&coloredVertexIDs_d, n_free_vertices * sizeof(int)));

			createColoring(mesh, n_free_vertices, &coloredVertexIDs, colorOffset_);

			gpuErrchk(hipMemcpyAsync(coloredVertexIDs_d, coloredVertexIDs, n_free_vertices * sizeof(int), hipMemcpyHostToDevice));
		}
#pragma omp section 
		{
			copyOpenMeshData(mesh, points, vertices, oneRingVec);
		}
	}

	gpuErrchk(hipMemcpyAsync(points_d, points, 2 * mesh.n_vertices() * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(vertices_d, vertices, n_free_vertices * sizeof(Vertex), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(oneRingVec_d, oneRingVec, oneRingVecLength * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk( hipDeviceSynchronize() );

	int n_colors = (int)colorOffset_.size() - 1;


	// face vector (only needed for quality evaluation)
#if PRINT_QUALITY
	int* faceVec = new int[mesh.n_faces() * 3];
	for (int i = 0; i < mesh.n_faces(); ++i) {
		TriMesh::FaceHandle fh = mesh.face_handle(i);
		int vertex_counter = 0;
		for (auto fv_it = mesh.fv_iter(fh); fv_it.is_valid(); ++fv_it) {
			faceVec[3 * i + vertex_counter++] = fv_it->idx();
		}
	}
	int* faceVec_d;
	gpuErrchk(hipMalloc((void**)&faceVec_d, 3 * mesh.n_faces() * sizeof(int)));
	gpuErrchk(hipMemcpy(faceVec_d, faceVec, 3 * mesh.n_faces() * sizeof(int), hipMemcpyHostToDevice));
#endif

	const float affineFactor_ = 1.f / (float)(NQ - 1);


#if PRINT_QUALITY
	// q_min_vec for printing
	float *q_min_vec, *q_avg_vec;
	hipMallocManaged(&q_min_vec, (n_iter + 1) * sizeof(float));
	hipMallocManaged(&q_avg_vec, (n_iter + 1) * sizeof(float));

	printf("    ");
	for (int i = 0; i < N_QUALITY_COLS; ++i) {
		printf("<%1.3f|", (float)(i + 1) / (float)N_QUALITY_COLS);
	}
	printf("\n\n");
	printFaceQuality << <1, 1 >> >(vertexPos_d, faceVec_d, mesh.n_faces(), 3, q_crit);
	printFaceQuality << <1, 1 >> >(vertexPos_d, faceVec_d, mesh.n_faces(), 3, q_crit, q_min_vec, q_avg_vec);
#endif // PRINT_QUALITY

	//hipDeviceSynchronize();
	//sw.start();
	for (int i = 0; i < n_iter; ++i) {
		for (int cid = 0; cid < n_colors; ++cid) {
			const int nBlocks = colorOffset_[cid + 1] - colorOffset_[cid];
			const int nThreads = NQ * NQ / 2;
			optimizeHierarchical << <nBlocks, nThreads >> >(coloredVertexIDs_d, colorOffset_[cid], vertices_d, points_d, oneRingVec_d, affineFactor_, element_size, q_crit, grid_scale);
			gpuErrchk( hipDeviceSynchronize() );
		}
#if PRINT_QUALITY
		gpuErrchk( hipDeviceSynchronize() );
		printFaceQuality << <1, 1 >> >(vertexPos_d, faceVec_d, mesh.n_faces(), 3, q_crit);
		printFaceQuality << <1, 1 >> >(vertexPos_d, faceVec_d, mesh.n_faces(), 3, q_crit, q_min_vec, q_avg_vec);
#endif // PRINT_QUALITY
	}
	//hipDeviceSynchronize();
	//sw.stop();
	//std::cout << "DMO runtime: " << sw.runtimeStr() << std::endl;

#if PRINT_QUALITY
	gpuErrchk( hipDeviceSynchronize() );
	std::string ofs_name = "../output.txt";
	std::ofstream ofs(ofs_name);
	for (int i = 0; i < n_iter + 1; ++i) {
		ofs << i << " " << q_min_vec[i] << " " << q_avg_vec[i] << std::endl;
	}
	ofs.close();
#endif // PRINT_QUALITY
	gpuErrchk( hipDeviceSynchronize() );
	hipMemcpy(points, points_d, 2 * mesh.n_vertices() * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(points_d);
	hipFree(vertices_d);
	hipFree(oneRingVec_d);
	hipFree(coloredVertexIDs_d);

	delete[] vertices;
	delete[] oneRingVec;
	delete[] coloredVertexIDs;

#if PRINT_QUALITY
	delete[] faceVec;

	hipFree(faceVec_d);
	hipFree(q_min_vec);
	hipFree(q_avg_vec);
#endif // PRINT_QUALITY

	//sw.start();
	// write vertex positions back to mesh
	for (auto v_it = mesh.vertices_begin(); v_it != mesh.vertices_end(); ++v_it) {
		int id = v_it->idx();
		TriMesh::Point p = { points[2 * id], points[2 * id + 1], 0.f };
		mesh.set_point(*v_it, p);
	}
	//sw.stop();
	//std::cout << "Write back runtime: " << sw.runtimeStr() << std::endl;

	delete[] points;
}

/////////////////////////////////////////////////////
//////////////////// CPU Version ////////////////////
inline void optimizeHierarchical(const int vid, const Vertex* vertices, float* points, int* oneRingVec, const float affineFactor_, const int element_size, const int q_crit, const float grid_scale) {

	const Vertex& v = vertices[vid];

	float xPos, yPos;
	float maxDistx = 0, maxDisty = 0;

	float oneRing[MAX_ONE_RING_SIZE];

	// min/max search + loading oneRing
	for (int k = 0; k < v.n_oneRing - 1; ++k) {
		float oneRingX = points[2 * oneRingVec[v.oneRingID + k]];
		float oneRingY = points[2 * oneRingVec[v.oneRingID + k] + 1];
		oneRing[2 * k] = oneRingX;
		oneRing[2 * k + 1] = oneRingY;

		float xDist = abs(points[2 * v.id] - oneRingX);
		float yDist = abs(points[2 * v.id + 1] - oneRingY);

		maxDistx = fmaxf(maxDistx, xDist);
		maxDisty = fmaxf(maxDisty, yDist);
	}

	// set xmaxmin...
	maxDistx = grid_scale * maxDistx;
	maxDisty = grid_scale * maxDisty;

	oneRing[2 * v.n_oneRing - 2] = points[2 * oneRingVec[v.oneRingID + v.n_oneRing - 1]];
	oneRing[2 * v.n_oneRing - 1] = points[2 * oneRingVec[v.oneRingID + v.n_oneRing - 1] + 1];

	xPos = points[2 * v.id];
	yPos = points[2 * v.id + 1];

	float pOld[2] = { xPos, yPos };
	float q = quality(v.n_oneRing, oneRing, pOld, element_size, q_crit);

	// start depth iteration
	float depth_scale = grid_scale;
	for (int depth = 0; depth < DEPTH; ++depth) {

		float xMax, xMin, yMax, yMin;
		xMax = xPos + depth_scale * maxDistx;
		xMin = xPos - depth_scale * maxDistx;
		yMax = yPos + depth_scale * maxDisty;
		yMin = yPos - depth_scale * maxDisty;

#pragma omp parallel for
		for (int i = 0; i < NQ; ++i) {
			float pos_i = affineFactor_ * (i * xMin + (NQ - 1 - i) * xMax);

			for (int j = 0; j < NQ; ++j) {
				float pos_j = affineFactor_ * (j * yMin + (NQ - 1 - j) * yMax);

				float pCurrent[2] = { pos_i, pos_j };
				float qCurrent = quality(v.n_oneRing, oneRing, pCurrent, element_size, q_crit);

				#pragma omp critical
				if (qCurrent > q) {
					xPos = pos_i;
					yPos = pos_j;
					q = qCurrent;
				}
			}
		}

		//depth dependent scaling factor
		depth_scale = depth_scale * (2.f / (NQ - 1));
	}


	// set new position if it is better than the old one
	points[2 * v.id] = xPos;
	points[2 * v.id + 1] = yPos;
}

void discreteMeshOptimizationCPU(TriMesh& mesh, const int q_crit = Q_MEANRATIO, const float grid_scale = 0.5f, int n_iter = 100) {

	int n_free_vertices = 0;
	for (auto v_it = mesh.vertices_begin(); v_it != mesh.vertices_end(); ++v_it) { if (!mesh.is_boundary(*v_it)) ++n_free_vertices; }
	//printf("N free vertices = %d\n", n_free_vertices);

	int oneRingVecLength = 0;
	for (auto v_it = mesh.vertices_begin(); v_it != mesh.vertices_end(); ++v_it) {
		if (mesh.is_boundary(*v_it)) { continue; }
		for (auto vv_it = mesh.vv_iter(*v_it); vv_it.is_valid(); ++vv_it) { ++oneRingVecLength; }
		++oneRingVecLength;		// additional count s.th. last element is again the first element
	}

	// convert OpenMesh to a basic structure
	float* points = new float[2 * mesh.n_vertices()];
	Vertex* vertices = new Vertex[n_free_vertices];
	int* oneRingVec = new int[oneRingVecLength];

	Stopwatch sw;
	copyOpenMeshData(mesh, points, vertices, oneRingVec);

	const float affineFactor_ = 1.f / (float)(NQ - 1);

	//sw.start();
	for (int i = 0; i < n_iter; ++i) {
		for (int vid = 0; vid < n_free_vertices; ++vid) {
			optimizeHierarchical(vid, vertices, points, oneRingVec, affineFactor_, 3, q_crit, grid_scale);
		}
	}
	//sw.stop();
	//std::cout << "DMO runtime: " << sw.runtimeStr() << std::endl;

	// write vertex positions back to mesh
	for (auto v_it = mesh.vertices_begin(); v_it != mesh.vertices_end(); ++v_it) {
		int id = v_it->idx();
		TriMesh::Point p = { points[2 * id], points[2 * id + 1], 0.f };
		mesh.set_point(*v_it, p);
	}

	delete[] points;
	delete[] vertices;
	delete[] oneRingVec;
}


template void discreteMeshOptimization( TriMesh&, const int, const float, int );
template void discreteMeshOptimization( PolyMesh&, const int, const float, int );